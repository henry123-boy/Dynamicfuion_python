//  ================================================================
//  Created by Gregory Kramida (https://github.com/Algomorph) on 6/28/23.
//  Copyright (c) 2023 Gregory Kramida
//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at

//  http://www.apache.org/licenses/LICENSE-2.0

//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
//  ================================================================
#include "core/linalg/TransposeBlocks.h"
#include <open3d/utility/Logging.h>
namespace utility = open3d::utility;

#ifdef BUILD_CUDA_MODULE

// third-party includes
#include <open3d/core/Dispatch.h>
#include <open3d/core/CUDAUtils.h>

// local includes
#include "core/linalg/LinalgKernels.cuh"
#include "core/linalg/PointerAggregationForBatchOperationsCUDA.cuh"


namespace o3c = open3d::core;


namespace nnrt::core::linalg::internal {


template<typename TElement>
inline void TransposeBlocksInPlaceCUDA_TypeDispatched(open3d::core::Tensor& blocks) {
	auto device = blocks.GetDevice();
	int64_t block_size = blocks.GetShape(1);
	int64_t block_count = blocks.GetShape(0);
	o3c::AssertTensorShape(blocks, { block_count, block_size, block_size });

	auto block_data = blocks.GetDataPtr<TElement>();

	TElement** block_array_device;
	auto size_of_pointer_array = block_count * sizeof(TElement*);
	OPEN3D_CUDA_CHECK(hipMalloc(&block_array_device, size_of_pointer_array));

	internal::GetMatrixPointersFromContiguousArrayOfMatrices_CUDA(block_array_device, block_data, block_size, block_size, block_count, device);
	internal::transpose_batched_cuda<TElement>(block_array_device, block_count, block_size, device);

	OPEN3D_CUDA_CHECK(hipFree(block_array_device));
}

void TransposeBlocksInPlaceCUDA(open3d::core::Tensor& blocks) {
	DISPATCH_DTYPE_TO_TEMPLATE(
			blocks.GetDtype(),
			[&] {
				TransposeBlocksInPlaceCUDA_TypeDispatched<scalar_t>(blocks);
			}
	);
}
#else
void TransposeBlocksInPlaceCUDA(open3d::core::Tensor& blocks) {
	utility::LogError("Attempting to call TransposeBlocksInPlaceCUDA routine when library not compiled with BUILD_CUDA_MODULE=ON");
}
#endif

} // namespace nnrt::core::linalg::interla